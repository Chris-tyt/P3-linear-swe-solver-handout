#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>

#include "../common/common.hpp"
#include "../common/solver.hpp"

// Here we hold the number of cells we have in the x and y directions
int nx, ny;

// This is where all of our points are. We need to keep track of our active
// height and velocity grids, but also the corresponding derivatives. The reason
// we have 2 copies for each derivative is that our multistep method uses the
// derivative from the last 2 time steps.
double *h, *u, *v, *dh, *du, *dv, *dh1, *du1, *dv1, *dh2, *du2, *dv2;
double H, g, dx, dy, dt;

// GPU device pointers
double *gpu_h, *gpu_u, *gpu_v;
double *gpu_dh, *gpu_du, *gpu_dv;
double *gpu_dh1, *gpu_du1, *gpu_dv1;
double *gpu_dh2, *gpu_du2, *gpu_dv2;

/**
 * This is your initialization function! We pass in h0, u0, and v0, which are
 * your initial height, u velocity, and v velocity fields. You should send these
 * grids to the GPU so you can do work on them there, and also these other fields.
 * Here, length and width are the length and width of the domain, and nx and ny are
 * the number of grid points in the x and y directions. H is the height of the water
 * column, g is the acceleration due to gravity, and dt is the time step size.
 * The rank and num_procs variables are unused here, but you will need them
 * when doing the MPI version.
 */
void init(double *h0, double *u0, double *v0, double length_, double width_, int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_)
{
    // @TODO: your code here
    // TODO: Your code here
    // We set the pointers to the arrays that were passed in
    h = h0;
    u = u0;
    v = v0;

    nx = nx_;
    ny = ny_;

    H = H_;
    g = g_;

    dx = length_ / nx;
    dy = width_ / nx;

    dt = dt_;

    // We allocate memory for the derivatives
    dh = (double *)calloc(nx * ny, sizeof(double));
    du = (double *)calloc(nx * ny, sizeof(double));
    dv = (double *)calloc(nx * ny, sizeof(double));

    dh1 = (double *)calloc(nx * ny, sizeof(double));
    du1 = (double *)calloc(nx * ny, sizeof(double));
    dv1 = (double *)calloc(nx * ny, sizeof(double));

    dh2 = (double *)calloc(nx * ny, sizeof(double));
    du2 = (double *)calloc(ny * ny, sizeof(double));
    dv2 = (double *)calloc(nx * ny, sizeof(double));

    // Allocate GPU memory for h, u, v and their derivatives
    hipMalloc((void **)&gpu_h, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_u, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_v, nx * ny * sizeof(double));

    // Transfer data from host to GPU
    hipMemcpy(gpu_h, h0, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_u, u0, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_v, v0, nx * ny * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void **)&gpu_dh, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_du, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_dv, nx * ny * sizeof(double));

    hipMalloc((void **)&gpu_dh1, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_du1, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_dv1, nx * ny * sizeof(double));

    hipMalloc((void **)&gpu_dh2, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_du2, nx * ny * sizeof(double));
    hipMalloc((void **)&gpu_dv2, nx * ny * sizeof(double));
}

void swap_buffers()
{
    double *gpu_tmp;

    gpu_tmp = gpu_dh2;
    gpu_dh2 = gpu_dh1;
    gpu_dh1 = gpu_dh;
    gpu_dh = gpu_tmp;

    gpu_tmp = gpu_du2;
    gpu_du2 = gpu_du1;
    gpu_du1 = gpu_du;
    gpu_du = gpu_tmp;

    gpu_tmp = gpu_dv2;
    gpu_dv2 = gpu_dv1;
    gpu_dv1 = gpu_dv;
    gpu_dv = gpu_tmp;
    
    double *tmp;

    tmp = dh2;
    dh2 = dh1;
    dh1 = dh;
    dh = tmp;

    tmp = du2;
    du2 = du1;
    du1 = du;
    du = tmp;

    tmp = dv2;
    dv2 = dv1;
    dv1 = dv;
    dv = tmp;
}

__global__ void compute_ghost_horizontal_gpu(double *h, int nx, int ny)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < ny; j += stride)
    {
        h(nx, j) = h(0, j);
    }
}

__global__ void compute_ghost_vertical_gpu(double *h, int nx, int ny)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < nx; i += stride)
    {
        h(i, ny) = h(i, 0);
    }
}

__global__ void compute_dh_gpu(double *dh, double *u, double *v, double dx, double dy, int nx, int ny, double H)
{
    // get index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // get stride
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int x = i; x < nx; x += stride_x)
    {
        for (int y = j; y < ny; y += stride_y)
        {
            dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
        }
    }
}

__global__ void compute_du_gpu(double *du, double *h, double dx, double dy, int nx, int ny, double g)
{
    // get index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // get stride
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int x = i; x < nx; x += stride_x)
    {
        for (int y = j; y < ny; y += stride_y)
        {
            du(i, j) = -g * dh_dx(i, j);
        }
    }
}

__global__ void compute_dv_gpu(double *dv, double *h, double dx, double dy, int nx, int ny, double g)
{
    // get index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // get stride
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = blockDim.y * gridDim.y;

    for (int x = i; x < nx; x += stride_x)
    {
        for (int y = j; y < ny; y += stride_y)
        {
            dv(i, j) = -g * dh_dy(i, j);
        }
    }
}

__global__ void compute_boundaries_horizontal_gpu(double *u, int nx, int ny)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int j = index; j < ny; j += stride)
    {
        u(0, j) = u(nx, j);
    }
}

__global__ void compute_boundaries_vertical_gpu(double *v, int nx, int ny)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < nx; i += stride)
    {
        v(i, 0) = v(i, ny);
    }
}

int t = 0;

/**
 * This is your step function! Here, you will actually numerically solve the shallow
 * water equations. You should update the h, u, and v fields to be the solution after
 * one time step has passed.
 */
void step()
{
    int bs = 256;
    int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

    dim3 bs_2(16, 16);
    dim3 sms_2(numSMs * 32, numSMs * 32);

    // First
    hipMemcpy(gpu_h, h, nx * ny * sizeof(double), hipMemcpyHostToDevice);

    // compute_ghost_horizontal
    compute_ghost_horizontal_gpu<<<32 * numSMs, bs>>>(gpu_h, nx, ny);

    // compute_ghost_vertical
    compute_ghost_vertical_gpu<<<32 * numSMs, bs>>>(gpu_h, nx, ny);

    // Next, compute the derivatives of fields
    // compute_dh
    hipMemcpy(gpu_dh, dh, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_du, du, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_dv, dv, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_u, u, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_v, v, nx * ny * sizeof(double), hipMemcpyHostToDevice);

    compute_dh_gpu<<<sms_2, bs_2>>>(gpu_dh, gpu_u, gpu_v, dx, dy, nx, ny, H);
    compute_du_gpu<<<sms_2, bs_2>>>(gpu_du, gpu_h, dx, dy, nx, ny, g);
    compute_dv_gpu<<<sms_2, bs_2>>>(gpu_dv, gpu_h, dx, dy, nx, ny, g);

    hipMemcpy(dh, gpu_dh, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(du, gpu_du, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(dv, gpu_dv, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h, gpu_h, nx * ny * sizeof(double), hipMemcpyDeviceToHost);


    // We set the coefficients for our multistep method
    double a1, a2, a3;

    if (t == 0)
    {
        a1 = 1.0;
    }
    else if (t == 1)
    {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    }
    else
    {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }

    // Finally, compute the next time step using multistep method
    // multistep(a1, a2, a3);
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
            u(i + 1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
            v(i, j + 1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
        }
    }

    // We compute the boundaries for our fields, as they are (1) needed for
    // the next time step, and (2) aren't explicitly set in our multistep method
    hipMemcpy(gpu_u, u, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(gpu_v, v, nx * ny * sizeof(double), hipMemcpyHostToDevice);
    // compute_boundaries_horizontal
    compute_boundaries_horizontal_gpu<<<32 * numSMs, bs>>>(gpu_u, nx, ny);

    // compute_boundaries_vertical
    compute_boundaries_horizontal_gpu<<<32 * numSMs, bs>>>(gpu_v, nx, ny);
    
    hipMemcpy(u, gpu_u, nx * ny * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(v, gpu_v, nx * ny * sizeof(double), hipMemcpyDeviceToHost);

    // We swap the buffers for our derivatives so that we can use the derivatives
    // from the previous time steps in our multistep method, then increment
    // the time step counter
    swap_buffers();

    t++;
}

/**
 * This is your transfer function! You should copy the h field back to the host
 * so that the CPU can check the results of your computation.
 */
void transfer(double *h_host)
{
    // @TODO: Your code here
    return;
}

/**
 * This is your finalization function! You should free all of the memory that you
 * allocated on the GPU here.
 */
void free_memory()
{
    // Free GPU memory
    hipFree(gpu_h);
    hipFree(gpu_u);
    hipFree(gpu_v);

    hipFree(gpu_dh);
    hipFree(gpu_du);
    hipFree(gpu_dv);

    hipFree(gpu_dh1);
    hipFree(gpu_du1);
    hipFree(gpu_dv1);

    hipFree(gpu_dh2);
    hipFree(gpu_du2);
    hipFree(gpu_dv2);

    // TODO: Your code here
    free(dh);
    free(du);
    free(dv);

    free(dh1);
    free(du1);
    free(dv1);

    free(dh2);
    free(du2);
    free(dv2);
}