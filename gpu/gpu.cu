#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>

#include "../common/common.hpp"
#include "../common/solver.hpp"


// Here we hold the number of cells we have in the x and y directions
int nx, ny;

// This is where all of our points are. We need to keep track of our active
// height and velocity grids, but also the corresponding derivatives. The reason
// we have 2 copies for each derivative is that our multistep method uses the
// derivative from the last 2 time steps.
double *h, *u, *v, *dh, *du, *dv, *dh1, *du1, *dv1, *dh2, *du2, *dv2;
double H, g, dx, dy, dt;


/**
 * This is your initialization function! We pass in h0, u0, and v0, which are
 * your initial height, u velocity, and v velocity fields. You should send these
 * grids to the GPU so you can do work on them there, and also these other fields.
 * Here, length and width are the length and width of the domain, and nx and ny are
 * the number of grid points in the x and y directions. H is the height of the water
 * column, g is the acceleration due to gravity, and dt is the time step size.
 * The rank and num_procs variables are unused here, but you will need them
 * when doing the MPI version.
 */
void init(double *h0, double *u0, double *v0, double length_, double width_, int nx_, int ny_, double H_, double g_, double dt_, int rank_, int num_procs_)
{
    // @TODO: your code here
        // TODO: Your code here
    // We set the pointers to the arrays that were passed in
    h = h0;
    u = u0;
    v = v0;

    nx = nx_;
    ny = ny_;

    // We allocate memory for the derivatives
    dh = (double *)calloc(nx * ny, sizeof(double));
    du = (double *)calloc(nx * ny, sizeof(double));
    dv = (double *)calloc(nx * ny, sizeof(double));

    dh1 = (double *)calloc(nx * ny, sizeof(double));
    du1 = (double *)calloc(nx * ny, sizeof(double));
    dv1 = (double *)calloc(nx * ny, sizeof(double));

    dh2 = (double *)calloc(nx * ny, sizeof(double));
    du2 = (double *)calloc(ny * ny, sizeof(double));
    dv2 = (double *)calloc(nx * ny, sizeof(double));

    H = H_;
    g = g_;

    dx = length_ / nx;
    dy = width_ / nx;

    dt = dt_;
}

void swap_buffers()
{
    double *tmp;

    tmp = dh2;
    dh2 = dh1;
    dh1 = dh;
    dh = tmp;

    tmp = du2;
    du2 = du1;
    du1 = du;
    du = tmp;

    tmp = dv2;
    dv2 = dv1;
    dv1 = dv;
    dv = tmp;
}

int t = 0;


/**
 * This is your step function! Here, you will actually numerically solve the shallow
 * water equations. You should update the h, u, and v fields to be the solution after
 * one time step has passed.
 */
void step()
{
    // @TODO: Your code here
    // First
    // compute_ghost_horizontal
    for (int j = 0; j < ny; j++)
    {
        h(nx, j) = h(0, j);
    }
    // compute_ghost_vertical
    for (int i = 0; i < nx; i++)
    {
        h(i, ny) = h(i, 0);
    }

    // Next, compute the derivatives of fields
    // compute_dh
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            dh(i, j) = -H * (du_dx(i, j) + dv_dy(i, j));
        }
    }
    // compute_du
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            du(i, j) = -g * dh_dx(i, j);
        }
    }
    // compute_dv
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            dv(i, j) = -g * dh_dy(i, j);
        }
    }

    // We set the coefficients for our multistep method
    double a1, a2, a3;

    if (t == 0)
    {
        a1 = 1.0;
    }
    else if (t == 1)
    {
        a1 = 3.0 / 2.0;
        a2 = -1.0 / 2.0;
    }
    else
    {
        a1 = 23.0 / 12.0;
        a2 = -16.0 / 12.0;
        a3 = 5.0 / 12.0;
    }

    // Finally, compute the next time step using multistep method
    // multistep(a1, a2, a3);
    for (int i = 0; i < nx; i++)
    {
        for (int j = 0; j < ny; j++)
        {
            h(i, j) += (a1 * dh(i, j) + a2 * dh1(i, j) + a3 * dh2(i, j)) * dt;
            u(i + 1, j) += (a1 * du(i, j) + a2 * du1(i, j) + a3 * du2(i, j)) * dt;
            v(i, j + 1) += (a1 * dv(i, j) + a2 * dv1(i, j) + a3 * dv2(i, j)) * dt;
        }
    }

    // We compute the boundaries for our fields, as they are (1) needed for
    // the next time step, and (2) aren't explicitly set in our multistep method
    // compute_boundaries_horizontal
    for (int j = 0; j < ny; j++)
    {
        u(0, j) = u(nx, j);
    }
    // compute_boundaries_vertical
    for (int i = 0; i < nx; i++)
    {
        v(i, 0) = v(i, ny);
    }

    // We swap the buffers for our derivatives so that we can use the derivatives
    // from the previous time steps in our multistep method, then increment
    // the time step counter
    swap_buffers();

    t++;
}

/**
 * This is your transfer function! You should copy the h field back to the host
 * so that the CPU can check the results of your computation.
 */
void transfer(double *h_host)
{
    // @TODO: Your code here
    return;
}

/**
 * This is your finalization function! You should free all of the memory that you
 * allocated on the GPU here.
 */
void free_memory()
{
    // @TODO: Your code here
    // TODO: Your code here
    free(dh);
    free(du);
    free(dv);

    free(dh1);
    free(du1);
    free(dv1);

    free(dh2);
    free(du2);
    free(dv2);
}